/*
Ravishka Rathnasuriya
GPU Project - 4th order Runge- Kutta algorithm
shared memory version for 1024 data sets using NVIDIA gtx device

Running the jobs
Copy the file RavishkaShared1024.c and shared1024 files
in your winscp folder.
using putty, go to the folder that contains above two files.
using:
"sbatch shared1024" command you can execute the program
and the .out file will be made in your folder with results. 
*/
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define size 1024
#define threadsize 1024


//function prototypes
//for global memory and it will call the device diffOfy
__global__ void rk4thOrder(double *gx0,double *gx, double *gh, double *gy);
__device__ double diffOfy(double x, double y);

int main(){
    //data sets for x, x0, and h values
 double xsamples[8] = {0.3, 0.6,0.8,1.2, 1.5, 1.6,1.8,2.0};
 double x0samples[8] = {0.1, 0.2,0.4,0.3,0.5,0.4,0.6,0.5};
 double hsamples[8] = {0.1,0.2,0.4,0.3,0.3,0.4,0.3,0.5};
 

 //declaring memory for x0,x,h,y values dynamically

 double *x0 = (double *)malloc(size *sizeof(double));
 double *x = (double *)malloc(size *sizeof(double));
 double *h = (double *)malloc(size *sizeof(double));
 double *y = (double *)malloc(size *sizeof(double));

 //total bytes
 const int totalsize = size*sizeof(double);
 //declaring memory for global memory variables 
 double *Gx0; double *Gx; double *Gh;double *Gy; 
 hipMalloc((void**)&Gx0,totalsize);
 hipMalloc((void**)&Gx,totalsize);
 hipMalloc((void**)&Gh,totalsize);
 hipMalloc((void**)&Gy,totalsize);


   //for timing the intervals
   hipEvent_t start, stop;
   //create two events start and stop
   hipEventCreate(&start);
   hipEventCreate(&stop);


//assigning values
//from 8th element onwards it will store same values
//for testing purposes.
 for(int i = 0; i< size; i++){
     if (i < 8){
         x[i] = xsamples[i];
         x0[i] = x0samples[i];
         h[i] = hsamples[i];
     }else{
     x[i]= 0.4;
     x0[i]= 0.0;
     h[i] = 0.1;
     }
 }

 //copying memory from host to device
//from x0,x, and h values to global memory
 hipMemcpy(Gx0, x0, totalsize, hipMemcpyHostToDevice);
 hipMemcpy(Gx, x, totalsize, hipMemcpyHostToDevice);
 hipMemcpy(Gh, h, totalsize, hipMemcpyHostToDevice);
 //cudaMemcpy(Gy, y, totalsize, cudaMemcpyHostToDevice);
 
//allocating number of blocks and threads in a block
 dim3 dimGrid(size/threadsize,1,1);
 dim3 dimBlock(threadsize,1,1);

  //start the time interval
hipEventRecord(start);
rk4thOrder<<< dimGrid, dimBlock>>>(Gx0,Gx,Gh,Gy);
//end the timing
hipEventRecord(stop);
//copy memory from device back to host 
hipMemcpy(y, Gy, totalsize, hipMemcpyDeviceToHost);

//wait until all device codes executes
hipEventSynchronize(stop);
float milliseconds = 0.0;
//store the time difference in variable milliseconds
hipEventElapsedTime(&milliseconds,start,stop);

//compute time in milli seconds,
printf("elapsed time is %lf milli secs \n",milliseconds);
//print the resulting values
for(int k = 0 ; k <size; k++){
printf("Answer at %dth position for dif eq is %.6lf \n",k, y[k]);
}
//freeing the memory 
hipFree(Gx0);
hipFree(Gx);
hipFree(Gh);
hipFree(Gh);


}


/*function rk4thoder - global function which is in the device
//parameters x0,y0,x,h,y 
return type double

the function will call diff eq function and compute the 4th order dif equation
iteration = (x -x0 /h)
k1, k2, k3, k4 will call diffofy function and iterate each of the time iteration results.*/

__global__
void rk4thOrder(double *gx0, double *gx, double *gh, double *gy){
   //invoking shared memory for xo,x,h,y variables. 
        double sx0[threadsize];
        double sx[threadsize];
        double sh[threadsize];
        double sy[threadsize];

    //copying data from global to shared memory using each iteration. 
        for(int m = 0; m < size/threadsize; m++){
         sx0[threadIdx.x] = gx0[m*threadsize + threadIdx.x];
         sx[threadIdx.x] = gx[m*threadsize + threadIdx.x];
        sh[threadIdx.x] = gh[m*threadsize + threadIdx.x];
         sy[threadIdx.x] = gy[m*threadsize + threadIdx.x];
         __syncthreads(); //wait till everything is done 

         //computing iteration numbers
        int iteration = ((sx[threadIdx.x]- sx0[threadIdx.x])/sh[threadIdx.x]);
       
        double k1, k2,k3,k4;
        sy[threadIdx.x] = 0.0;
        //for above iterations it will compute k1, k2,k3, k4 values and store in our new y value
        for(int j = 1; j <= iteration; j++){
            k1 = sh[threadIdx.x] * diffOfy(sx0[threadIdx.x],sy[threadIdx.x]);
            k2 = sh[threadIdx.x] * diffOfy((sx0[threadIdx.x]+ sh[threadIdx.x]/2), (sy[threadIdx.x] + k1/2));
            k3 = sh[threadIdx.x] * diffOfy((sx0[threadIdx.x] + sh[threadIdx.x]/2), (sy[threadIdx.x]+k2/2));
            k4 = sh[threadIdx.x]* diffOfy((sx0[threadIdx.x]+sh[threadIdx.x]), (sy[threadIdx.x]+k3));
            sy[threadIdx.x] += ((1.0/6.0)*(k1 +2*k2+2*k3+k4));
            sx0[threadIdx.x] += sh[threadIdx.x];
        }
        //copy the results from shared memory back to global memory
        gy[ m*threadsize + threadIdx.x] = sy[threadIdx.x];
        __syncthreads();
    }
}


//function name diffofy - device function
//parameters x, y
//return type double
//used a simple function that returns square of x and y summation
__device__
double diffOfy(double x, double y){
    return ((x*x)+ (y*y));
}