/*
Ravishka Rathnasuriya
GPU Project - 4th order Runge- Kutta algorithm
GPU version for 1024 data sets using NVIDIA gtx device

Running the jobs
Copy the file RavishkaGPU1024.c and gpu1024 files
in your winscp folder.
using putty, go to the folder that contains above two files.
using:
"sbatch gpu1024" command you can execute the program
and the .out file will be made in your folder with results. 
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define size 1024
#define threadsize 1024


//function prototypes
//for global memory and it will call the device diffOfy
__global__ void rk4thOrder(double *gx0,double *gx, double *gh, double *gy);
__device__ double diffOfy(double x, double y);

int main(){

    //data sets for x, x0, and h values
 double xsamples[8] = {0.3, 0.6,0.8,1.2, 1.5, 1.6,1.8,2.0};
 double x0samples[8] = {0.1, 0.2,0.4,0.3,0.5,0.4,0.6,0.5};
 double hsamples[8] = {0.1,0.2,0.4,0.3,0.3,0.4,0.3,0.5};
 

 //declaring memory for x0,x,h,y values dynamically

 double *x0 = (double *)malloc(size *sizeof(double));
 double *x = (double *)malloc(size *sizeof(double));
 double *h = (double *)malloc(size *sizeof(double));
 double *y = (double *)malloc(size *sizeof(double));

 //total bytes
 const int totalsize = size*sizeof(double);
 //declaring memory for global memory variables 
 double *Gx0; double *Gx; double *Gh;double *Gy; 
 hipMalloc((void**)&Gx0,totalsize);
 hipMalloc((void**)&Gx,totalsize);
 hipMalloc((void**)&Gh,totalsize);
 hipMalloc((void**)&Gy,totalsize);


   //for timing the intervals
   hipEvent_t start, stop;
   //create two events start and stop
   hipEventCreate(&start);
   hipEventCreate(&stop);

//assigning values
//from 8th element onwards it will store same values
//for testing purposes. 

 for(int i = 0; i< size; i++){
     if (i < 8){
         x[i] = xsamples[i];
         x0[i] = x0samples[i];
         h[i] = hsamples[i];
     }else{
     x[i]= 0.4;
     x0[i]= 0.0;
     h[i] = 0.1;
     }
 }
//copying memory from host to device
//from x0,x, and h values to global memory
 hipMemcpy(Gx0, x0, totalsize, hipMemcpyHostToDevice);
 hipMemcpy(Gx, x, totalsize, hipMemcpyHostToDevice);
 hipMemcpy(Gh, h, totalsize, hipMemcpyHostToDevice);
 //cudaMemcpy(Gy, y, totalsize, cudaMemcpyHostToDevice);
 
//allocating number of blocks and threads in a block
 dim3 dimGrid(size/threadsize,1,1);
 dim3 dimBlock(threadsize,1,1);

 //start tge time interval
hipEventRecord(start);
rk4thOrder<<< dimGrid, dimBlock>>>(Gx0,Gx,Gh,Gy);
//end the timing
hipEventRecord(stop);
hipMemcpy(y, Gy, totalsize, hipMemcpyDeviceToHost);

//wait until all device codes executes
hipEventSynchronize(stop);
float milliseconds = 0.0;
//store the time difference in variable milliseconds
hipEventElapsedTime(&milliseconds,start,stop);

//compute time in milli seconds,
printf("elapsed time is %lf milli secs \n",milliseconds);
//print the resulting values
for(int k = 0 ; k <size; k++){
    printf("Answer at %dth position for dif eq is %.6lf \n",k, y[k]);
}
  //freeing the memory  
hipFree(Gx0);
hipFree(Gx);
hipFree(Gh);
hipFree(Gh);
}


/*function rk4thoder - global function which is in the device
//parameters x0,y0,x,h,y 
return type double

the function will call diff eq function and compute the 4th order dif equation
iteration = (x -x0 /h)
k1, k2, k3, k4 will call diffofy function and iterate each of the time iteration results.*/

__global__
void rk4thOrder(double *gx0, double *gx, double *gh, double *gy){
    int i = threadIdx.x + blockIdx.x* blockDim.x;
        int iteration = ((gx[i]- gx0[i])/gh[i]); //computing iteration numbers
       
        double k1, k2,k3,k4;
        gy[i] = 0.0;
       
         //for above iterations it will compute k1, k2,k3, k4 values and store in our new y value
        for(int j = 1; j <= iteration; j++){
            k1 = gh[i] * diffOfy(gx0[i],gy[i]);
            k2 = gh[i] * diffOfy((gx0[i]+ gh[i]/2), (gy[i] + k1/2));
            k3 = gh[i] * diffOfy((gx0[i] + gh[i]/2), (gy[i]+k2/2));
            k4 = gh[i]* diffOfy((gx0[i]+gh[i]), (gy[i]+k3));
            gy[i] += ((1.0/6.0)*(k1 +2*k2+2*k3+k4));
            gx0[i] += gh[i];
        }
    

}

//function name diffofy - device function
//parameters x, y
//return type double
//used a simple function that returns square of x and y summation
__device__
double diffOfy(double x, double y){
    return ((x*x)+ (y*y));
}